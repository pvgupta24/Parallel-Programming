#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

//Catch Cuda errors
void catchCudaError(hipError_t error){
    if(error!=hipSuccess) {
        printf("\n====== Cuda Error Code %i ======\n %s\n",error,hipGetErrorString(error)); 
        exit(-1); 
    }
}
//=====================================================================

#define DIM 32

#define ROW1 40
#define COMMON_WIDTH 30
#define COL2 4000

//Kernel function
__global__ void multiply(float a[][COMMON_WIDTH], float b[][COL2], float c[][COL2]){
    //Skip till required block + the required thread index in the block
    uint x = blockDim.x * blockIdx.x + threadIdx.x;
    uint y = blockDim.y * blockIdx.y + threadIdx.y;

    float cell = 0;
    if(x < ROW1 && y < COL2){
        for(uint i = 0; i < COMMON_WIDTH; ++i)
            cell += a[x][i]*b[i][y];
        c[x][y] = cell;//c has ROW1 x COL2 dim.
    }
}

int main(){

    float a[ROW1][COMMON_WIDTH], b[COMMON_WIDTH][COL2], c[ROW1][COL2]; //Host 2-d arrays
    float (*d_a)[COMMON_WIDTH], (*d_b)[COL2], (*d_c)[COL2]; //Device 2-d arrays

    clock_t start, end;
    hipEvent_t d_start, d_end;
    catchCudaError(hipEventCreate(&d_start));
    catchCudaError(hipEventCreate(&d_end));

    size_t sizeA = ROW1 * COMMON_WIDTH * sizeof(float);
    size_t sizeB = COMMON_WIDTH * COL2 * sizeof(float);
    size_t sizeC = ROW1 * COL2 * sizeof(float);
    //Allocate device memory(double ptr as assigning value to a pointer as defined in CUDA API)
    catchCudaError(hipMalloc((void **)&d_a, sizeA));
    catchCudaError(hipMalloc((void **)&d_b, sizeB));
    catchCudaError(hipMalloc((void **)&d_c, sizeC));

    //Initial values of a,b random
    for(uint i=0; i < ROW1; ++i)
        for(uint j=0; j < COMMON_WIDTH; ++j)
            a[i][j] = i+j;

    for(uint i=0; i < COMMON_WIDTH; ++i)
        for(uint j=0; j < COL2; ++j)
            b[i][j] = i-j;    
    

    //Copy to Device
    catchCudaError(hipMemcpy(d_a, a, sizeA, hipMemcpyHostToDevice));
    catchCudaError(hipMemcpy(d_b, b, sizeB, hipMemcpyHostToDevice));

    catchCudaError(hipEventRecord(d_start));

    dim3 dimGrid(DIM, DIM);
    dim3 dimBlock(ceil(1.0*ROW1/DIM), ceil(1.0*COL2/DIM)) ;
    //Max 1024 threads in each block(max 65,535)
    multiply <<< dimGrid, dimBlock >>>(d_a, d_b, d_c);
    catchCudaError(hipEventRecord(d_end));    
    
    //Copy to Host
    catchCudaError(hipMemcpy(c, d_c, sizeC, hipMemcpyDeviceToHost));

    //Wait for all threads to finish
    //catchCudaError(cudaDeviceSynchronize());

    //Waits till event is recorded
    catchCudaError(hipEventSynchronize(d_end));
    float cell;
    start = clock();
    for(uint i=0; i<ROW1; ++i)
        for(uint j=0; j<COL2; ++j){
            cell = 0;        
            for(uint k=0; k<COMMON_WIDTH; ++k)
                cell += a[i][k]*b[k][j];
        }
    end = clock();
    float time_taken = 1000.0* (end - start)/CLOCKS_PER_SEC;
    float d_time_taken;
    hipEventElapsedTime(&d_time_taken, d_start, d_end);

    printf("Host time = %f ms\nDevice Time = %f ms\n", time_taken, d_time_taken);    
    //Free device memory
    catchCudaError(hipFree(d_a));
    catchCudaError(hipFree(d_b));
    catchCudaError(hipFree(d_c));

}   

/*
Output
Correct matrix multiplication
Host time = 11.943000 ms
Device Time = 0.252992 ms
*/
